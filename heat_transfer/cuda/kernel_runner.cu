#include "common.h"
#include "kernel.cu"
#include "heat_grid.h"

// Threads per each dimension (x and y)
const int THREADS_PER_DIM = 16;

void evolve_heat(HeatGrid *grid, float speed, int time_steps) {
    int x_blocks = (grid->w + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
    int y_blocks = (grid->h + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

    dim3 blocks(x_blocks, y_blocks);
    dim3 threads(THREADS_PER_DIM, THREADS_PER_DIM);

    // Start heat evolution
    CUDA_CHECK(hipEventRecord(grid->start, 0));
    bool dstOut = true;
    for (int i = 0; i < time_steps * 2; i++) {
        float *in, *out;
        if (dstOut) {
            in = grid->dev_inSrc;
            out = grid->dev_outSrc;
        } else {
            in = grid->dev_outSrc;
            out = grid->dev_inSrc;
        }
        // Copy information about heaters in the grid
        copy_const_kernel<<<blocks, threads>>>(in, grid->w, grid->h);
        blend_kernel<<<blocks, threads>>>(out, dstOut, speed, grid->w, grid->h);
        dstOut = !dstOut;
    }
    CUDA_CHECK(hipMemcpy(grid->host_outSrc, grid->dev_inSrc, grid->float_grid_n_bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(grid->stop, 0));
    CUDA_CHECK(hipEventSynchronize(grid->stop));
    grid->measure_elapsed_time();
    grid->frames += time_steps;
}


void bind_grid(HeatGrid* grid) {
    size_t null_offset = 0;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(
        &null_offset,
        texConstSrc,                            // The texture ptr
        grid->dev_constSrc, desc, grid->w, grid->h,   // Info about the buffer to be binded
        sizeof(float) * grid->w                // Pitch? Not sure what it means
    ));
    CUDA_CHECK(hipBindTexture2D(
        &null_offset,
        texIn,                                  // The texture ptr
        grid->dev_inSrc, desc, grid->w, grid->h,      // Info about the buffer to be binded
        sizeof(float) * grid->w                // Pitch? Not sure what it means
    ));
    CUDA_CHECK(hipBindTexture2D(
        &null_offset,
        texOut,                                 // The texture ptr
        grid->dev_outSrc, desc, grid->w, grid->h,     // Info about the buffer to be binded
        sizeof(float) * grid->w                // Pitch? Not sure what it means
    ));
    printf("Successfully binded grid.\n");
}


void unbind_grid() {
    CUDA_CHECK(hipUnbindTexture(texIn));
    CUDA_CHECK(hipUnbindTexture(texOut));
    CUDA_CHECK(hipUnbindTexture(texConstSrc));
}
